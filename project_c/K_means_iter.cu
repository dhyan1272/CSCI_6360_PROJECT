#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#define X 16384  //X dimession of the data 
#define Y 3		//Y dimesnion of the data //TODO NEED TO KNOW THE DATA SIZE BEFORE IMPORTING
#define K 2		//NUMBER OF CLUSTERS TO DIVIDE THE DATA INTO
#define MAX_ITERS 5  //NUMBER OF ITERATIONS

double *num=NULL;
double *centroids_c=NULL;
int *idx=NULL;

__global__
void findclosestcentroids(double* num, double* centroids_c, int* idx){

	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread (number enters the loop the next time, as the thread id will be same)
	for(int i=index; i<X; i+=stride){
		
		int x=index+offset*stride;
		int j, l, min_ind; 
		double sum, dist[K],min_dist;
		
		for (j=0;j<K;j++){
			
			sum=0;
			for (l=0;l<Y;l++){

					sum=sum+(*(num+x*Y+l)-*(centroids_c+j*Y+l))*(*(num+x*Y+l)-*(centroids_c+j*Y+l));

			}
			dist[j]=sqrt(sum);
		}
		min_dist=dist[0];
		min_ind=0;
		for (j=0; j<K; j++){
			
			if (dist[j]<min_dist) {

				min_dist=dist[j];
				min_ind=j;

			}
		}
		*(idx+x)=min_ind;
		offset++;
	}
	
}

__global__
void computeCentroids(double* num, int* idx, double* centroids_c){

	
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread (number enters the loop the next time, as the thread id will be same)
	
	int  m, j, l, count;
	double sum[Y]; 					//for(i=0;i<Y;i++) sum[i]=0.0;//is it reqd ?
	for(int i=index; i<K; i+=stride){

		int x=index+offset*stride;
		count=0;
		for(m=0;m<Y;m++) sum[m]=0.0;

		for(j =0; j<X; j++){

			if(idx[j]==x){

					count++;
					for (l=0;l<Y;l++){

						sum[l]=sum[l]+ *(num+j*Y+l);
					
					}
			
			}

		}
		//printf("Counts is %d \n", count);
		for (l=0;l<Y;l++){

			*(centroids_c+x*Y+l)=sum[l]/count;					
		}
	}

}

int main(){

	FILE *fp, *fw;

	int lower =0;
	int upper =X-1;
	srand(time(0));

	double num1;
	int i, j, k, n_blocks, no_of_threads,rnd_num;
	
	no_of_threads=32;
	if (no_of_threads==(X*Y))
        n_blocks = (X*Y)/no_of_threads; //calculation of number of blocks baseds on threadscounts and world size
    else 
        n_blocks = (X*Y)/no_of_threads+1;;

	//Initializing CUDA memory 
	hipMallocManaged(&num, sizeof(double)*X*Y);
	hipMallocManaged(&centroids_c, sizeof(double)*K*Y);
	hipMallocManaged(&idx, sizeof(int)*X);

	//Opening file and loading data into CUDA memory.
	fp=fopen("input.txt","r");
	if(fp==NULL) {
		printf("Exiting no file with such name \n");
		exit(-1);
	}
	//Loading the 2-dimensional data into a 1D VECTOR for CUDA. 
	for (i=0;i<X;i++){
		for (j=0;j<Y;j++){
			fscanf(fp,"%lf", &num1);
			num[i*Y+j]=num1;
			//printf(" %.15lf ", num1);  //Just for debugging
		}
		//printf("\n");
	}
	fclose(fp);

	for (i = 0; i < K; i++) {

			rnd_num = (rand()%(upper-lower + 1)) + lower;
			//printf("%d ", rnd_num);  
			for (j=0;j<Y;j++){ 
        		*(centroids_c+i*Y+j) = *(num+rnd_num*Y+j);
        		//printf("Centroids are %e",*(centroids_c+i*Y+j)); 
        	} 
        //printf("\n");
    }

	for (i=0;i<MAX_ITERS;i++){

		int cudaDeviceCount;
		hipError_t cE1,cE2;
		findclosestcentroids<<< n_blocks, no_of_threads>>>(num, centroids_c, idx);
		cE1=hipGetDeviceCount( &cudaDeviceCount);
		cE2=hipDeviceSynchronize();
		//printf("The rwo error is %d %d \n",cE1,cE2);
		const char* x_err=hipGetErrorString (cE2);
		//printf("%s \n",x_err); 

		computeCentroids<<<1, 32>>>(num, &idx[0], centroids_c);
		hipDeviceSynchronize();

	}

	/*
	for(i=0; i<K;i++){
			for(j=0; j<Y;j++){

				printf("Centroids using CUDA %lf  ",*(centroids_c+i*Y+j));

			}
		printf("\n");
	}
	*/

	for (i=0; i<X;i++){
		//printf("%d==%d\n",i+1, idx[i]+1);

		for (k=0;k<K;k++){

			if (idx[i]==k){

					for (j=0;j<Y;j++){			
						*(num+i*Y+j)=*(centroids_c+k*Y+j);
					}
			}
				
		}

	}
	fw=fopen("output.txt","w");
	
	for(i=0; i<X;i++){
	
		for(j=0; j<Y;j++){

				fprintf(fw,"%lf  ",*(num+i*Y+j));
				//	printf("%lf  ",num[i][j]);
			}
		fprintf(fw, "\n");
		//printf("\n");
	}

	fclose(fw);
	hipFree(num);
	hipFree(centroids_c);
	hipFree(idx);
	return 0;

}
