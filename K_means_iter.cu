#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#include<time.h>
#define X 300  //X dimession of the data 
#define Y 2		//Y dimesnion of the data //TODO NEED TO KNOW THE DATA SIZE BEFORE IMPORTING
#define K 3		//NUMBER OF CLUSTERS TO DIVIDE THE DATA INTO
#define MAX_ITERS 1  //NUMBER OF ITERATIONS
#define blockSize 32

double *num=NULL;
double *centroids_c=NULL;
int *idx=NULL;
double *sum=NULL;
int *count=NULL;

__global__
void findclosestcentroids(double* num, double* centroids_c, int* idx){

	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread (number enters the loop the next time, as the thread id will be same)
	for(int i=index; i<X*Y; i+=stride){
		
		int x=index+offset*stride;
		int j, l, min_ind; 
		double sum, dist[K],min_dist;
		
		for (j=0;j<K;j++){
			sum=0;
			for (l=0;l<Y;l++){
					sum=sum+(*(num+x*Y+l)-*(centroids_c+j*Y+l))*(*(num+x*Y+l)-*(centroids_c+j*Y+l));
			}
			dist[j]=sqrt(sum);
		}
		min_dist=dist[0];
		min_ind=0;
		for (j=0; j<K; j++){
			if (dist[j]<min_dist) {
				min_dist=dist[j];
				min_ind=j;
			}
		}
		*(idx+x)=min_ind;
		offset++;
	}
}

__global__ void reduce(double *g_idata, double *g_odata, int *g_odata_count, const unsigned int m, const int *idx, const unsigned int cl)
{
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockSize*2) + tid;
  if (idx[i]==cl) {
    unsigned int gridSize = blockSize*2*gridDim.x;
    const unsigned int n = X*Y;
    __shared__ double sdata[n];
    __shared__ double sdata_count[n];

    sdata[tid] = 0.0;
    sdata_count[tid] = 0;

    while (i < n) {
      sdata[tid] += g_idata[(i*Y+m)] + g_idata[(i*Y+m)+blockSize];
      sdata_count[tid] += 2; 
      i += gridSize;
    }
    __syncthreads();

    if (blockSize >= 512) {if (tid < 256) { sdata[tid] += sdata[tid + 256]; sdata_count[tid] += sdata_count[tid + 256] ;} __syncthreads(); }
    if (blockSize >= 256) {if (tid < 128) { sdata[tid] += sdata[tid + 128]; sdata_count[tid] += sdata_count[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) {if (tid < 64) { sdata[tid] += sdata[tid +   64]; sdata_count[tid] += sdata_count[tid +   64]; } __syncthreads(); }
    
    if (tid < 32) {
        if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
        if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
        if (blockSize >=  16) sdata[tid] += sdata[tid +  8];
        if (blockSize >=  8) sdata[tid] += sdata[tid +  4];
        if (blockSize >=  4) sdata[tid] += sdata[tid +  2];
        if (blockSize >=  2) sdata[tid] += sdata[tid +  1];

        if (blockSize >=  64) sdata_count[tid] += sdata_count[tid + 32];
        if (blockSize >=  32) sdata_count[tid] += sdata_count[tid + 16];
        if (blockSize >=  16) sdata_count[tid] += sdata_count[tid +  8];
        if (blockSize >=  8) sdata_count[tid] += sdata_count[tid +  4];
        if (blockSize >=  4) sdata_count[tid] += sdata_count[tid +  2];
        if (blockSize >=  2) sdata_count[tid] += sdata_count[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
    if (tid == 0) g_odata_count[blockIdx.x] = sdata_count[0];
  }
}

void computeCentroids(double* num, int* idx, double* centroids, int n_blocks) {
  double sum_g = 0.0;
  int count_g = 0;
  hipMallocManaged(&sum, sizeof(double)*n_blocks);
  hipMallocManaged(&count, sizeof(int)*n_blocks);
  for (int i=0; i<K; i++) {
    for (int m=0; m<Y; m++) {
      for(int j=0; j <n_blocks; ++j) {
	sum[j] = 0.0;
	count[j] = 0;
      }
      reduce <<<n_blocks, blockSize>>> (num, sum, count, m, idx, i);
      hipDeviceSynchronize();

      //do blocksum
      sum_g = 0.0; count_g = 0;
      for(int j=0; j <n_blocks; ++j) {
	sum_g += sum[j];
	count_g += count[j];
      }
      printf("m = %d, sum=%f, count=%d \n", m, sum_g, count_g);
      *(centroids+i*Y+m)=sum_g/count_g;
    }
  }
}

int main(){
	/*To generate random centroids
        srand(time(0)); 
	for(int i = 0; i<K; i++) {
	  for(int i = 0; i<Y; i++) centroids[i][j] = rand()%10;
	}
	*/

	double centroids[K][Y]={{3,3},{6,2},{8,5}};
	double num1;
	int i, j, n_blocks;
	n_blocks = (X*Y + blockSize - 1)/blockSize;
/*
	if (no_of_threads==(X*Y))
        n_blocks = (X*Y)/no_of_threads; //calculation of number of blocks baseds on threadscounts and world size
    else 
        n_blocks = (X*Y)/no_of_threads+1;;
*/
	//Initializing CUDA memory 
	hipMallocManaged(&num, sizeof(double)*X*Y);
	hipMallocManaged(&centroids_c, sizeof(double)*K*Y);
	hipMallocManaged(&idx, sizeof(int)*X);

	//Opening file and loading data into CUDA memory.
	FILE *fp;
	fp=fopen("data.txt","r");
	if(fp==NULL) {
		printf("Exiting no file with such name \n");
		exit(-1);
	}
	//Loading the 2-dimensional data into a 1D VECTOR for CUDA. 
	for (i=0;i<X;i++){
		for (j=0;j<Y;j++){
			fscanf(fp,"%lf", &num1);
			num[i*Y+j]=num1;
		}
	}
	fclose(fp);

	//Loading the 2-dimensional centroid Initialization data into a 1D VECTOR for CUDA. 
	for (i=0;i<K;i++){
		for (j=0;j<Y;j++){
			centroids_c[i*Y+j]=centroids[i][j];
		}
	}

	for (i=0;i<MAX_ITERS;i++){
//question: why are we passing the global arrays as arguments??, the functions already have the info
		findclosestcentroids <<<n_blocks, blockSize>>> (num, &centroids_c[0], &idx[0]);
		hipDeviceSynchronize();
		computeCentroids(num, &idx[0], &centroids_c[0], n_blocks);
	}
	
	//for (i=0;i<X;i++){
	//	printf("%d===%d \n",i+1, idx[i]+1);
	//}
	return 0;
}
