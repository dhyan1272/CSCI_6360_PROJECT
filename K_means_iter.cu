#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define X 300  //X dimession of the data 
#define Y 2		//Y dimesnion of the data //TODO NEED TO KNOW THE DATA SIZE BEFORE IMPORTING
#define K 3		//NUMBER OF CLUSTERS TO DIVIDE THE DATA INTO
#define MAX_ITERS 1  //NUMBER OF ITERATIONS

double *num=NULL;
double *centroids_c=NULL;
int *idx=NULL;

__global__
void findclosestcentroids(double* num, double* centroids_c, int* idx){

	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread (number enters the loop the next time, as the thread id will be same)
	for(int i=index; i<X*Y; i+=stride){
		
		int x=index+offset*stride;
		int j, l, min_ind; 
		double sum, dist[K],min_dist;
		
		for (j=0;j<K;j++){
			
			sum=0;
			for (l=0;l<Y;l++){

					sum=sum+(*(num+x*Y+l)-*(centroids_c+j*Y+l))*(*(num+x*Y+l)-*(centroids_c+j*Y+l));

			}
			dist[j]=sqrt(sum);
			printf("Distance of %d %e\n", index, sum);
		}
		min_dist=dist[0];
		min_ind=0;
		for (j=0; j<K; j++){
			
			if (dist[j]<min_dist) {

				min_dist=dist[j];
				min_ind=j;

			}
		}
		*(idx+x)=min_ind;
		offset++;
	}
	
}

__global__
void computeCentroids(double* num, int* idx, double* centroids_c){

	
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread (number enters the loop the next time, as the thread id will be same)
	
	int i, m, j, l, count;
	double sum[Y]; 					//for(i=0;i<Y;i++) sum[i]=0.0;//is it reqd ?
	for(int i=index; i<K; i+=stride){

		int x=index+offset*stride;
		count=0;
		for(m=0;m<Y;m++) sum[m]=0.0;

		for(j =0; j<X; j++){

			if(idx[j]==x){

					count++;
					for (l=0;l<Y;l++){

						sum[l]=sum[l]+ *(num+j*Y+l);
					
					}
			
			}

		}
		printf("COunts is %d \n", count);
		for (l=0;l<Y;l++){

			*(centroids_c+x*Y+l)=sum[l]/count;					
		}
	}

}

int main(){

	FILE *fp;

	//initialization TODO make it random
	double centroids[K][Y]={{3,3},{6,2},{8,5}};

	double num1;
	int i, j, n_blocks, no_of_threads;
	
	no_of_threads=32;
	if (no_of_threads==(X*Y))
        n_blocks = (X*Y)/no_of_threads; //calculation of number of blocks baseds on threadscounts and world size
    else 
        n_blocks = (X*Y)/no_of_threads+1;;

	//Initializing CUDA memory 
	hipMallocManaged(&num, sizeof(double)*X*Y);
	hipMallocManaged(&centroids_c, sizeof(double)*K*Y);
	hipMallocManaged(&idx, sizeof(int)*X);

	//Opening file and loading data into CUDA memory.
	fp=fopen("data.txt","r");
	if(fp==NULL) {
		printf("Exiting no file with such name \n");
		exit(-1);
	}
	//Loading the 2-dimensional data into a 1D VECTOR for CUDA. 
	for (i=0;i<X;i++){
		for (j=0;j<Y;j++){
			fscanf(fp,"%lf", &num1);
			num[i*Y+j]=num1;
			//printf(" %.15lf ", num1);  //Just for debugging
		}
		//printf("\n");
	}
	fclose(fp);

	//Loading the 2-dimensional centroid Initialization data into a 1D VECTOR for CUDA. 
	for (i=0;i<K;i++){
		for (j=0;j<Y;j++){
			centroids_c[i*Y+j]=centroids[i][j];
			//printf(" %.15lf ", centroids_c[i*Y+j]);  //Just for debugging
		}
		//printf("\n");
	}
	for (i=0;i<MAX_ITERS;i++){

		findclosestcentroids<<< n_blocks, no_of_threads>>>(num, &centroids_c[0], &idx[0]);
		hipDeviceSynchronize();
		computeCentroids<<<1,32>>>(num, &idx[0], &centroids_c[0]);
		hipDeviceSynchronize();

	}
	
	for (i=0;i<X;i++){

		printf("%d===%d \n",i+1, idx[i]+1);
	}

	for(i=0; i<K;i++){
			for(j=0; j<Y;j++){

				printf("Centroids using CUDA %lf  ",*(centroids_c+i*Y+j));

			}
		printf("\n");
	}
	return 0;

}
