#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#include<time.h>
#define X 300  //X dimession of the data 
#define Y 2		//Y dimesnion of the data //TODO NEED TO KNOW THE DATA SIZE BEFORE IMPORTING
#define K 3		//NUMBER OF CLUSTERS TO DIVIDE THE DATA INTO
#define MAX_ITERS 1  //NUMBER OF ITERATIONS
#define blockSize 32

double *num = NULL;
double *centroids_c = NULL;
int *idx = NULL;
double *sum = NULL;
int *count = NULL;
double *num_in_clust = NULL;
int *count_in_clust = NULL;

__global__
void findclosestcentroids(double* num, double* centroids_c, int* idx, double* num_in_clust, int* count_in_clust){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int stride=blockDim.x*gridDim.x;
	int offset=0; //offset keeps track if the same thread (number enters the loop the next time, as the thread id will be same)
	for(int i=index; i<X*Y; i+=stride){
		int x=index+offset*stride;
		int j, l, min_ind; 
		double sum_l, dist[K],min_dist;
		
		for (j=0;j<K;j++){
			sum_l=0;
			for (l=0;l<Y;l++){
					sum_l = sum_l+(*(num+x*Y+l)-*(centroids_c+j*Y+l))*(*(num+x*Y+l)-*(centroids_c+j*Y+l));
			}
			dist[j]=sqrt(sum_l);
		}
		min_dist=dist[0];
		min_ind=0;
		for (j=0; j<K; j++){
			if (dist[j]<min_dist) {
				min_dist=dist[j];
				min_ind=j;
			}
		}
		*(idx+x) = min_ind;
		for (l=0;l<Y;l++){
	          *(num_in_clust+(min_ind*X*Y)+x*Y+l) = *(num+x*Y+l);
	          *(count_in_clust+(min_ind*X*Y)+x*Y+l) = 1;
		}
		offset++;
	}
}

__global__ void reduce(double *g_idata, int* g_idata_count, double *g_odata, int *g_odata_count, const unsigned int m, const int *idx, const unsigned int cl)
{
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockSize*2) + tid;
//  if (idx[i]==cl) {
    unsigned int gridSize = blockSize*2*gridDim.x;
    const unsigned int n = X*Y;
    __shared__ double sdata[n];
    __shared__ double sdata_count[n];

    sdata[tid] = 0.0;
    sdata_count[tid] = 0;

    while (i < n) {
      sdata[tid] += *(g_idata+(i*Y+m)) + *(g_idata+(i*Y+m)+blockSize);
      sdata_count[tid] += *(g_idata_count+(i*Y+m)) + *(g_idata_count+(i*Y+m)+blockSize); 
      //sdata_count[tid] += 2; 
      i += gridSize;
    }
    __syncthreads();

    if (blockSize >= 512) {if (tid < 256) { sdata[tid] += sdata[tid + 256]; sdata_count[tid] += sdata_count[tid + 256] ;} __syncthreads(); }
    if (blockSize >= 256) {if (tid < 128) { sdata[tid] += sdata[tid + 128]; sdata_count[tid] += sdata_count[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) {if (tid < 64) { sdata[tid] += sdata[tid + 64]; sdata_count[tid] += sdata_count[tid + 64]; } __syncthreads(); }
    
    if (tid < 32) {
        if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
        if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
        if (blockSize >=  16) sdata[tid] += sdata[tid +  8];
        if (blockSize >=  8) sdata[tid] += sdata[tid +  4];
        if (blockSize >=  4) sdata[tid] += sdata[tid +  2];
        if (blockSize >=  2) sdata[tid] += sdata[tid +  1];

        if (blockSize >=  64) sdata_count[tid] += sdata_count[tid + 32];
        if (blockSize >=  32) sdata_count[tid] += sdata_count[tid + 16];
        if (blockSize >=  16) sdata_count[tid] += sdata_count[tid +  8];
        if (blockSize >=  8) sdata_count[tid] += sdata_count[tid +  4];
        if (blockSize >=  4) sdata_count[tid] += sdata_count[tid +  2];
        if (blockSize >=  2) sdata_count[tid] += sdata_count[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
    if (tid == 0) g_odata_count[blockIdx.x] = sdata_count[0];
//  }
}

void computeCentroids(double* num, int* idx, double* centroids, int n_blocks, double* num_in_clust, int* count_in_clust) {
  double sum_g = 0.0;
  int count_g = 0;
  hipMallocManaged(&sum, sizeof(double)*n_blocks);
  hipMallocManaged(&count, sizeof(int)*n_blocks);
  for (int i=0; i<K; i++) {
    for (int m=0; m<Y; m++) {
      for(int j=0; j <n_blocks; ++j) {
	sum[j] = 0.0;
	count[j] = 0;
      }
      reduce <<<n_blocks, blockSize>>> (&num_in_clust[X*Y*i], &count_in_clust[X*Y*i], sum, count, m, idx, i);
      hipDeviceSynchronize();

      //do blocksum
      sum_g = 0.0; count_g = 0;
      for(int j=0; j <n_blocks; ++j) {
	sum_g += sum[j];
	count_g += count[j];
      }
      printf("k=%d, m=%d, centroid=%f \n", i, m, sum_g/count_g);
      *(centroids+i*Y+m)=sum_g/count_g;
    }
  }
}

int main(){
	/*To generate random centroids
        srand(time(0)); 
	for(int i = 0; i<K; i++) {
	  for(int i = 0; i<Y; i++) centroids[i][j] = rand()%10;
	}
	*/

	double centroids[K][Y]={{3,3},{6,2},{8,5}};
	double num1;
	int i, j, n_blocks;
	n_blocks = (X*Y + blockSize - 1)/blockSize;

	//Initializing CUDA memory 
	hipMallocManaged(&num, sizeof(double)*X*Y);
	hipMallocManaged(&centroids_c, sizeof(double)*K*Y);
	hipMallocManaged(&idx, sizeof(int)*X);
        hipMallocManaged(&num_in_clust, sizeof(double)*X*Y*K);
        hipMallocManaged(&count_in_clust, sizeof(int)*X*Y*K);

	//Opening file and loading data into CUDA memory.
	FILE *fp;
	fp=fopen("data.txt","r");
	if(fp==NULL) {
		printf("Exiting no file with such name \n");
		exit(-1);
	}
	//Loading the 2-dimensional data into a 1D VECTOR for CUDA. 
	for (i=0;i<X;i++){
		for (j=0;j<Y;j++){
			fscanf(fp,"%lf", &num1);
			num[i*Y+j]=num1;
		}
	}
	fclose(fp);

	//Loading the 2-dimensional centroid Initialization data into a 1D VECTOR for CUDA. 
	for (i=0;i<K;i++){
		for (j=0;j<Y;j++){
			centroids_c[i*Y+j]=centroids[i][j];
		}
	}

	for (i=0;i<MAX_ITERS;i++){
		findclosestcentroids <<<n_blocks, blockSize>>> (num, &centroids_c[0], &idx[0], &num_in_clust[0], &count_in_clust[0]);
		hipDeviceSynchronize();
		//for (j=0; j<K; ++j) printf("k=%d, count=%d\n", j, count_in_clust[j]);
		computeCentroids(num, &idx[0], &centroids_c[0], n_blocks, num_in_clust, count_in_clust);
	}
	return 0;
}
