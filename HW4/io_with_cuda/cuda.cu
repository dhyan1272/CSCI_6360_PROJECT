/*
* The code has been written by Karan Bhanot, Abolaji Adesoji, Aditya Joshi and Dhyanjyoti Nath.
*
* Some function definitions are referenced from
* sample code provided by Christopher D. Carothers, 
* provided as part of his class assignment of Parallel Computing 
* Spring 2020.
*/

// Include headers (including CUDA)
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>


// Buffer
extern long long *buf;

/*
* Returns the inialized buffer on CUDA
*/
extern "C" void getBuffer( int rank, int numranks, long long blocksize )
{
    // Check and assign the device for this MPI rank
	hipError_t cE;
	int cudaDeviceCount;

    // Check if enough devices are available
	if ((cE = hipGetDeviceCount(&cudaDeviceCount)) != hipSuccess) {
		printf("Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount);
        exit(-1);
	}

    // Set device given that it is available
	if ((cE = hipSetDevice(rank % cudaDeviceCount)) != hipSuccess) {
		printf(" Unable to have rank %d set to cuda device %d, error is %d \n", rank, (rank % cudaDeviceCount), cE);
        exit(-1);
	}
	// Assign memory to the buf variable
	hipMallocManaged(&buf, blocksize);
}
